

#include <hip/hip_runtime.h>
#include <stdio.h>
/* Global variable for returning result of computation */
__device__ float g_res;


/* CUDA kernel for dot product. Optimized for brevity
 */
__global__ void dotp_kernel(double xs[], double ys[]) {
    const unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;
    double mine = xs[idx] * ys[idx];
    atomicAdd(&g_res, mine);
}

inline void gpuAssert(hipError_t err, const char* file, int line) {
    if( err != hipSuccess ) {
        fprintf(stderr,"GPU error: %s %s %d\n", hipGetErrorString(err), file, line);
        exit(1);
    }
}
#define GPU_ASSERT(ans) {gpuAssert(ans,__FILE__,__LINE__);}

/* Execute kernel on GPU */
extern "C"
double calculate_dot_p(double xs[], double ys[], int n) {
    // Deeply magical constant
    const int block_size = 512; 
    // size padded to nearest multiple of block size
    int n_padded = ((n + block_size - 1)/block_size)*block_size;
    dim3 block(block_size);
    dim3 grid (n_padded/block_size);
    // Buffers for CUDA data
    double *buf_xs = NULL;
    double *buf_ys = NULL;
    // Allocate memory and copy to device
    GPU_ASSERT( hipMalloc( &buf_xs, n*sizeof(double) ) );
    GPU_ASSERT( hipMalloc( &buf_ys, n*sizeof(double) ) );
    GPU_ASSERT( hipMemcpy(buf_xs, xs, n*sizeof(double), hipMemcpyHostToDevice) );
    GPU_ASSERT( hipMemcpy(buf_ys, ys, n*sizeof(double), hipMemcpyHostToDevice) );
    // Execute kernel
    dotp_kernel<<<grid, block>>>(buf_xs, buf_ys);
    // Free buffers & obtain data
    hipFree( &buf_xs );
    hipFree( &buf_ys );
    float res;
    GPU_ASSERT( hipMemcpyFromSymbol(&res, HIP_SYMBOL(g_res), sizeof(res), 0, hipMemcpyDeviceToHost) );
    return res;
}
