#include "hip/hip_runtime.h"
#include "GCF.cu"
#include "../FFT/cufftshift.cu"

#include "threadFenceReduction_kernel.cuh"

extern "C" __global__ void reduce_512_odd(const hipDoubleComplex *g_idata, double *g_odata, unsigned int n) {
  reduceSinglePass_dev<512, false>(g_idata, g_odata, n);
}

extern "C" __global__ void normalize(
    double * normp
  , hipDoubleComplex v[257*257]
  ){
  normalize_kernel<128>(1.0 / (*normp), v);
}
