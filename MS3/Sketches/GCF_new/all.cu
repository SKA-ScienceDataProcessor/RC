#include "hip/hip_runtime.h"
#include "scale_complex_by_dbl.cuh"
#include "GCF.cu"
#include "../FFT/cufftshift.cu"

#include "threadFenceReduction_kernel.cuh"

extern "C" __global__ void reduce_512_e2(const hipDoubleComplex *g_idata, double *g_odata, unsigned int n) {
  retirementCount = 0;
  reduceSinglePass_dev<512, true>(g_idata, g_odata, n);
}

extern "C" __global__ void normalize(
    double * normp
  , hipDoubleComplex * v
  , int len
  ){
  scale_complex_by_dbl(1.0 / (*normp), v, len);
}
