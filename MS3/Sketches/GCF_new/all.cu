#include "hip/hip_runtime.h"
#include "scale_complex_by_dbl.cuh"
#include "GCF.cu"
#include "../FFT/cufftshift.cu"

#include "threadFenceReduction_kernel.cuh"

extern "C" __global__ void reduce_512_odd(const hipDoubleComplex *g_idata, double *g_odata, unsigned int n) {
  reduceSinglePass_dev<512, false>(g_idata, g_odata, n);
}

extern "C" __global__ void normalize(
    double * normp
  , hipDoubleComplex * v
  , int len
  ){
  scale_complex_by_dbl(1.0 / (*normp), v, len);
}
