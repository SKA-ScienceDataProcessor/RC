#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hip/hip_math_constants.h>

template <int max_half_support>
__device__
void ucs_common(hipDoubleComplex mesh[max_half_support * 2 + 1][max_half_support * 2 + 1]){
  const int
      x = blockIdx.x * blockDim.x + threadIdx.x
    , y = blockIdx.y * blockDim.y + threadIdx.y
    ;
  double
      sc = double(max_half_support)
    , xs = double(x - max_half_support) / sc
    , ys = double(y - max_half_support) / sc
    ;

  mesh[x][y].x = xs * xs + ys * ys;
  mesh[x][y].x = 0.0;
}


// test instantiation
template __device__
void ucs_common<256>(hipDoubleComplex mesh[513][513]);

template <int max_half_support>
__device__
void calc_inplace(
    hipDoubleComplex mesh[max_half_support * 2 + 1][max_half_support * 2 + 1]
  , double t2
  , double w
  ){
  const int
      x = blockIdx.x * blockDim.x + threadIdx.x
    , y = blockIdx.y * blockDim.y + threadIdx.y
    ;
  double
      r2 = mesh[x][y].x * t2 * t2
    , ph = w * (1.0 - sqrt(1.0 - r2))
    ;
  double s, c;
  sincos(2.0 * HIP_PI * ph, &s, &c);
  mesh[x][y].x = s;
  mesh[x][y].y = c;
}

// test instantiation
template __device__
void calc_inplace<256>(
    hipDoubleComplex mesh[513][513]
  , double t2
  , double w
  );

template <int max_half_support>
__device__
void calc(
    hipDoubleComplex dst[max_half_support * 2 + 1][max_half_support * 2 + 1]
  , hipDoubleComplex src[max_half_support * 2 + 1][max_half_support * 2 + 1]
  , double t2
  , double w
  ){
  const int
      x = blockIdx.x * blockDim.x + threadIdx.x
    , y = blockIdx.y * blockDim.y + threadIdx.y
    ;
  double
      r2 = src[x][y].x * t2 * t2
    , ph = w * (1.0 - sqrt(1.0 - r2))
    ;
  double s, c;
  sincos(2.0 * HIP_PI * ph, &s, &c);
  dst[x][y].x = s;
  dst[x][y].y = c;
}

// test instantiation
template __device__
void calc<256>(
    hipDoubleComplex dst[513][513]
  , hipDoubleComplex src[513][513]
  , double t2
  , double w
  );

template <
    int max_half_support
  , int oversample
  >
__device__
void copy_ucs_2_over(
    hipDoubleComplex dst[max_half_support * oversample * 2 + 1][max_half_support * oversample * 2 + 1]
  , hipDoubleComplex src[max_half_support * 2 + 1][max_half_support * 2 + 1]
  ) {
  const int
      dst_center = max_half_support * oversample
    , pad = dst_center - max_half_support
    , cut = dst_center + max_half_support
    ;
  const int
      x = blockIdx.x * blockDim.x + threadIdx.x
    , y = blockIdx.y * blockDim.y + threadIdx.y
    ;
  if (
       x < pad
    || x > cut
    || y < pad
    || y > cut
    ) dst[x][y] = {0.0, 0.0};
  else
      dst[x][y] = src[x - pad][y - pad];
  }

// test instantiation
template __device__
void copy_ucs_2_over<256,8>(
    hipDoubleComplex dst[4097][4097]
  , hipDoubleComplex src[513][513]
  );
