#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hip/hip_math_constants.h>

// Use symmetry
// Use max_half_support threads only
// Perhaps it's not very cache friendly, but it is
// very simple to perform work-distribution for this variant

template <int max_half_support>
__device__
void ucs_common(
    hipDoubleComplex mesh[max_half_support * 2 + 1][max_half_support * 2 + 1]
  , double t2
  ){
  const int
      x = blockIdx.x * blockDim.x + threadIdx.x
    , y = blockIdx.y * blockDim.y + threadIdx.y
    ;
  double
      sc = double(max_half_support)
    , xs = double(x) / sc * t2
    , ys = double(y) / sc * t2
    ;
  hipDoubleComplex r2 = make_hipDoubleComplex(xs * xs + ys * ys, 0.0);

  mesh[max_half_support - x][max_half_support - y] = r2;
  mesh[max_half_support - x][max_half_support + y] = r2;
  mesh[max_half_support + x][max_half_support - y] = r2;
  mesh[max_half_support + x][max_half_support + y] = r2;
}

// test instantiation
template __device__
void ucs_common<256>(hipDoubleComplex mesh[513][513], double t2);


template <int max_half_support>
__device__
void calc_inplace(
    hipDoubleComplex mesh[max_half_support * 2 + 1][max_half_support * 2 + 1]
  , double w
  ){
  const int
      x = blockIdx.x * blockDim.x + threadIdx.x
    , y = blockIdx.y * blockDim.y + threadIdx.y
    ;
  // mesh is symmetric, thus we need no recalc ph
  double ph = w * (1.0 - sqrt(1.0 - mesh[max_half_support - x][max_half_support - y].x));
  double s, c;
  sincos(2.0 * HIP_PI * ph, &s, &c);

  hipDoubleComplex res = make_hipDoubleComplex(c, -s); // to get rid of conj later

  mesh[max_half_support - x][max_half_support - y] = res;
  mesh[max_half_support - x][max_half_support + y] = res;
  mesh[max_half_support + x][max_half_support - y] = res;
  mesh[max_half_support + x][max_half_support + y] = res;
}

// test instantiation
template __device__
void calc_inplace<256>(
    hipDoubleComplex mesh[513][513]
  , double w
  );


template <int max_half_support>
__device__
void calc(
    hipDoubleComplex dst[max_half_support * 2 + 1][max_half_support * 2 + 1]
  , const hipDoubleComplex src[max_half_support * 2 + 1][max_half_support * 2 + 1]
  , double w
  ){
  const int
      x = blockIdx.x * blockDim.x + threadIdx.x
    , y = blockIdx.y * blockDim.y + threadIdx.y
    ;
  double ph = w * (1.0 - sqrt(1.0 - src[max_half_support - x][max_half_support - y].x));
  double s, c;
  sincos(2.0 * HIP_PI * ph, &s, &c);

  hipDoubleComplex res = make_hipDoubleComplex(c, -s); // to get rid of conj later

  dst[max_half_support - x][max_half_support - y] = res;
  dst[max_half_support - x][max_half_support + y] = res;
  dst[max_half_support + x][max_half_support - y] = res;
  dst[max_half_support + x][max_half_support + y] = res;
}

// test instantiation
template __device__
void calc<256>(
    hipDoubleComplex dst[513][513]
  , const hipDoubleComplex src[513][513]
  , double w
  );


template <
    int max_half_support
  , int oversample
  >
__device__
void copy_ucs_2_over(
    hipDoubleComplex dst[max_half_support * oversample * 2 + 1][max_half_support * oversample * 2 + 1]
  , const hipDoubleComplex src[max_half_support * 2 + 1][max_half_support * 2 + 1]
  ) {
  const int
      dst_center = max_half_support * oversample
    , pad = dst_center - max_half_support
#ifndef __SET_NULL_PADDING
    , cut = dst_center + max_half_support
#endif
    ;
  const int
      x = blockIdx.x * blockDim.x + threadIdx.x
    , y = blockIdx.y * blockDim.y + threadIdx.y
    ;
#ifndef __SET_NULL_PADDING
  if (
       x < pad
    || x > cut
    || y < pad
    || y > cut
    ) dst[x][y] = {0.0, 0.0};
  else
#endif
      dst[x][y] = src[x - pad][y - pad];
  }

// test instantiation
template __device__
void copy_ucs_2_over<256,8>(
    hipDoubleComplex dst[4097][4097]
  , const hipDoubleComplex src[513][513]
  );


template <
    int max_half_support
  , int oversample
  >
__device__
void cut_out(
    int supp
  , hipDoubleComplex * dst
  , const hipDoubleComplex src[max_half_support * oversample * 2 + 1][max_half_support * oversample * 2 + 1]
  ) {
  const int
      x = blockIdx.x * blockDim.x + threadIdx.x
    , y = blockIdx.y * blockDim.y + threadIdx.y
    ;
  const int off = (max_half_support - supp) * oversample;

  dst[x * supp + y] = src[off+x][off+y];
  }

// test instantiation
template __device__
void cut_out<256,8>(
    int supp
  , hipDoubleComplex * dst
  , const hipDoubleComplex src[4097][4097]
  );
