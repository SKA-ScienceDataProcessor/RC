#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hip/hip_math_constants.h>

// Use symmetry
// Use max_half_support threads only
// Perhaps it's not very cache friendly, but it is
// very simple to perform work-distribution for this variant

#define __SET_MAP                               \
  const int                                     \
      x = blockIdx.x * blockDim.x + threadIdx.x \
    , y = blockIdx.y * blockDim.y + threadIdx.y \
    , xl = max_half_support - x                 \
    , xr = max_half_support + x                 \
    , yl = max_half_support - y                 \
    , yr = max_half_support + y                 \
    ;


template <int max_half_support>
__device__ __inline__
void ucs_common(
    hipDoubleComplex mesh[max_half_support * 2 + 1][max_half_support * 2 + 1]
  , double t2
  ){
  __SET_MAP
  double
      t2_div_sc = t2 / double(max_half_support)
    , xs = double(x) * t2_div_sc
    , ys = double(y) * t2_div_sc
    ;
  hipDoubleComplex r2 = make_hipDoubleComplex(xs * xs + ys * ys, 0.0);

  mesh[xl][yl] = r2;
  mesh[xl][yr] = r2;
  mesh[xr][yl] = r2;
  mesh[xr][yr] = r2;
}

extern "C" __global__ void r2(hipDoubleComplex mesh[257][257], double t2) {
  ucs_common<128>(mesh, t2);
}


#if 0
template <int max_half_support>
__device__
void calc_inplace(
    cuDoubleComplex mesh[max_half_support * 2 + 1][max_half_support * 2 + 1]
  , double w
  ){
  __SET_MAP
  // mesh is symmetric, thus we need no recalc ph
  double ph = w * (1.0 - sqrt(1.0 - mesh[xl][yl].x));
  double s, c;
  sincos(2.0 * CUDART_PI * ph, &s, &c);

  cuDoubleComplex res = make_cuDoubleComplex(c, -s); // to get rid of conj later

  mesh[xl][yl] = res;
  mesh[xl][yr] = res;
  mesh[xr][yl] = res;
  mesh[xr][yr] = res;
}

// test instantiation
template __device__
void calc_inplace<128>(
    cuDoubleComplex mesh[257][257]
  , double w
  );
#endif


template <int max_half_support>
__device__ __inline__
void calc(
    hipDoubleComplex dst[max_half_support * 2 + 1][max_half_support * 2 + 1]
  , const hipDoubleComplex src[max_half_support * 2 + 1][max_half_support * 2 + 1]
  , double w
  ){
  __SET_MAP
  double ph = w * (1.0 - sqrt(1.0 - src[xl][yl].x));
  double s, c;
  sincos(2.0 * HIP_PI * ph, &s, &c);

  hipDoubleComplex res = make_hipDoubleComplex(c, -s); // to get rid of conj later

  dst[xl][yl] = res;
  dst[xl][yr] = res;
  dst[xr][yl] = res;
  dst[xr][yr] = res;
}

extern "C" __global__ void wkernff(
    hipDoubleComplex dst[257][257]
  , const hipDoubleComplex src[257][257]
  , double w
  ){
  calc<128>(dst, src, w);
}


template <
    int max_half_support
  , int oversample
  >
__device__ __inline__
void copy_ucs_2_over(
    hipDoubleComplex dst[(max_half_support * 2 + 1) * oversample][(max_half_support * 2 + 1) * oversample]
  , const hipDoubleComplex src[max_half_support * 2 + 1][max_half_support * 2 + 1]
  ){
  const int dst_center = (max_half_support * 2 + 1) * oversample / 2;
  __SET_MAP
  dst[dst_center - x][dst_center - y] = src[xl][yl];
  dst[dst_center - x][dst_center + y] = src[xl][yr];
  dst[dst_center + x][dst_center - y] = src[xr][yl];
  dst[dst_center + x][dst_center + y] = src[xr][yr];
}

extern "C" __global__  void copy_2_over(
    hipDoubleComplex dst[2056][2056]
  , const hipDoubleComplex src[257][257]
  ){
  copy_ucs_2_over<128,8>(dst, src);
}


#ifdef __SMALL_EXTRACT
template <
    int max_half_support
  , int oversample
  >
__device__ __inline__
void extract_over(
    int overx
  , int overy
  , cuDoubleComplex dst[max_half_support * 2 + 1][max_half_support * 2 + 1]
  , const cuDoubleComplex src[(max_half_support * 2 + 1) * oversample][(max_half_support * 2 + 1) * oversample]
  ) {
  __SET_MAP
  const int
      sxl = xl * oversample + overx
    , sxr = xr * oversample + overx
    , syl = yl * oversample + overy
    , syr = yr * oversample + overy
    ;
  dst[xl][yl] = src[sxl][syl];
  dst[xl][yr] = src[sxl][syr];
  dst[xr][yl] = src[sxr][syl];
  dst[xr][yr] = src[sxr][syr];
  }

extern "C" __global__ void wextract0(
    int overx
  , int overy
  , cuDoubleComplex dst[257][257]
  , const cuDoubleComplex src[2056][2056]
  ){
  extract_over<128,8>(overx, overy, dst, src);
}
#else
// We use 3rd grid dimension to cover oversample range
template <
    int max_half_support
  , int oversample
  >
__device__ __inline__
void transpose_over(
    hipDoubleComplex dst[oversample][oversample][max_half_support * 2 + 1][max_half_support * 2 + 1]
  , const hipDoubleComplex src[(max_half_support * 2 + 1) * oversample][(max_half_support * 2 + 1) * oversample]
  ) {
  __SET_MAP
  const int
      overx = blockIdx.z / oversample
    , overy = blockIdx.z % oversample
    , sxl = xl * oversample + overx
    , sxr = xr * oversample + overx
    , syl = yl * oversample + overy
    , syr = yr * oversample + overy
    ;
  dst[overx][overy][xl][yl] = src[sxl][syl];
  dst[overx][overy][xl][yr] = src[sxl][syr];
  dst[overx][overy][xr][yl] = src[sxr][syl];
  dst[overx][overy][xr][yr] = src[sxr][syr];
}

extern "C" __global__ void transpose_over0(
    hipDoubleComplex dst[8][8][257][257]
  , const hipDoubleComplex src[2056][2056]
  ){
  transpose_over<128,8>(dst, src);
}
#endif


//
__device__ static __inline__ hipDoubleComplex cuMulComplexByDouble(hipDoubleComplex v,
                                                             double y){
  return make_hipDoubleComplex ( v.x * y
                              , v.y * y
                              );
}

// The work-distribution scheme here is very different from those above (and below).
template <
    int max_half_support
  >
__device__ __inline__
void normalize_kernel(
    double norm
  , hipDoubleComplex v[(max_half_support * 2 + 1) * (max_half_support * 2 + 1)]
  ) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < (max_half_support * 2 + 1) * (max_half_support * 2 + 1)) v[x] = cuMulComplexByDouble(v[x], norm);
}


template <
    int max_half_support
  , int oversample
  >
__device__ __inline__
void cut_out(
    int half_supp
  , hipDoubleComplex * dst
  , const hipDoubleComplex src[max_half_support * 2 + 1][max_half_support * 2 + 1]
  ) {
  __SET_MAP
  const int supp = half_supp * 2 + 1;

  if (x > half_supp || y > half_supp) return;

  const int
      dxl = (half_supp - x) * supp
    , dxr = (half_supp + x) * supp
    , dyl = half_supp - y
    , dyr = half_supp + y
    ;

  dst[dxl + dyl] = src[xl][yl];
  dst[dxl + dyr] = src[xl][yr];
  dst[dxr + dyl] = src[xr][yl];
  dst[dxr + dyr] = src[xr][yr];
}

extern "C" __global__ void wextract1(
    int supp
  , hipDoubleComplex * dst
  , const hipDoubleComplex src[257][257]
  ){
  cut_out<128,8>(supp, dst, src);
}
