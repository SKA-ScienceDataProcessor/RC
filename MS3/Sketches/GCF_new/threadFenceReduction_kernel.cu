#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
  Copyright (C) 2015 Braam Research, LLC.
  This software contains source code provided by NVIDIA Corporation.
  It is modified to fulfill specific needs of GCF computations.
 */

#include "threadFenceReduction_kernel.cuh"

bool isPow2(unsigned int x)
{
    return ((x&(x-1))==0);
}

template <unsigned int blockSize, bool nIsPow2>
__global__
void reduceSinglePass(const hipDoubleComplex *g_idata, double *g_odata, unsigned int n){
  reduceSinglePass_dev<blockSize, nIsPow2>(g_idata, g_odata, n);
}

// blocksize is >= 64 always
extern "C"
void reduceSinglePass(int size, int threads, int blocks, const hipDoubleComplex *d_idata, double *d_odata)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    int smemSize = threads * sizeof(double);

    // choose which of the optimized versions of reduction to launch
    if (isPow2(size))
    {
        switch (threads)
        {
            case 512:
                reduceSinglePass<512, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 256:
                reduceSinglePass<256, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 128:
                reduceSinglePass<128, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 64:
                reduceSinglePass< 64, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;
        }
    }
    else
    {
        switch (threads)
        {
            case 512:
                reduceSinglePass<512, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 256:
                reduceSinglePass<256, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 128:
                reduceSinglePass<128, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 64:
                reduceSinglePass< 64, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;
        }
    }
}

/* How to use it
void getNumBlocksAndThreads(int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
    if (n == 1)
    {
        threads = 1;
        blocks = 1;
    }
    else
    {
        threads = (n < maxThreads*2) ? nextPow2(n / 2) : maxThreads;
        blocks = max(1, n / (threads * 2));
    }

    blocks = min(maxBlocks, blocks);
}
*/
