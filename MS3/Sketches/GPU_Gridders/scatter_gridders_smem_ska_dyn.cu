#include "hip/hip_runtime.h"
#define __DYN_GRID_SIZE
#include "common.h"
#include "metrix.h"
#include "atomic_add.h"
#include "OskarBinReader.h"

#include <math_functions.h>

static __inline__ __device__ void loadVisIntoSharedMem (
    const Double4c vis[]
  , const double3 uvw[]
  , Double4c vis_shared[]
  , int timesteps_x_channels
  ) {
  for (int i = threadIdx.x; i < timesteps_x_channels; i += blockDim.x) {
    // Add rotation
    #define __ROT_N_COPY(pol) vis_shared[i].##pol = rotw(vis[i].##pol, uvw[i].z);
    __ROT_N_COPY(XX)
    __ROT_N_COPY(XY)
    __ROT_N_COPY(YX)
    __ROT_N_COPY(YY)
  }
}

template <
    int over
  , bool do_mirror
  >
__inline__ __device__ void loadUVWIntoSharedMem (
    double scale
  , double wstep
  , int max_supp // needed for u v translation
  , const double3 uvw[]
  , Pregridded uvo_shared[]
  , int2 off_shared[]
  , int timesteps_x_channels
  , int grid_size
  ) {
  for (int i = threadIdx.x; i < timesteps_x_channels; i += blockDim.x) {
    // uvo_shared[i] is passed by reference and updated!
    pregridPoint<over, do_mirror>(scale, wstep, uvw[i], uvo_shared[i], grid_size);
    off_shared[i].x = uvo_shared[i].u % max_supp;
    off_shared[i].y = uvo_shared[i].v % max_supp;
  }
}

template <
    bool is_half_gcf
  >
__inline__ __device__
// grid must be initialized to 0s.
void gridKernel_scatter_kernel_small(
    int max_supp
    // For full-size GCF should be passed 0-centered,
    // i.e. with 0-index in the middle
  , const complexd * gcf[]
  , Double4c _grid[]
  , const Pregridded uvo_shared[]
  , const Double4c vis[]
  , const int2 off_shared[]
  , int myU
  , int myV
  , int timesteps_x_channels
  , int grid_size
  ) {
  __ACC(Double4c, grid, grid_size);
  complexd
      sumXX = {0, 0}
    , sumXY = {0, 0}
    , sumYX = {0, 0}
    , sumYY = {0, 0}
    ;
  int
      grid_point_u = 0
    , grid_point_v = 0;

  for (int i = 0; i < timesteps_x_channels; i++) {
    int myConvU, myConvV, myGridU, myGridV, u, v;
    u = uvo_shared[i].u;
    v = uvo_shared[i].v;
    myConvU = myU - off_shared[i].x;
    if (myConvU < 0) myConvU += max_supp;
    myConvV = myV - off_shared[i].y;
    if (myConvV < 0) myConvV += max_supp;
    // This translates points by max_supp/2
    // returning them back to normal (they were translates by -max_supp/2 before)
    myGridU = u + myConvU;
    myGridV = v + myConvV;

    int supp = uvo_shared[i].gcf_layer_supp;

    complexd supportPixel;
    #define __layeroff myConvU * supp + myConvV
    if (is_half_gcf) {
      int index = uvo_shared[i].gcf_layer_index;
      // Negative index indicates that original w was mirrored
      // and we shall negate the index to obtain correct
      // offset *and* conjugate the result.
      if (index < 0) {
        supportPixel = gcf[-index][__layeroff];
        supportPixel.y = - supportPixel.y;
      } else {
        supportPixel = gcf[index][__layeroff];
      }
    } else {
        supportPixel = gcf[uvo_shared[i].gcf_layer_index][__layeroff];
    }

    if (myGridU != grid_point_u || myGridV != grid_point_v) {
      atomicAdd(&grid[grid_point_u][grid_point_v], sumXX, sumXY, sumYX, sumYY);
        sumXX
      = sumXY
      = sumYX
      = sumYY
      = make_hipDoubleComplex(0.0, 0.0);
      grid_point_u = myGridU;
      grid_point_v = myGridV;
    }
    #define __ADD_SUPP(pol) sum##pol = hipCfma(supportPixel, vis[i].pol, sum##pol)
    __ADD_SUPP(XX);
    __ADD_SUPP(XY);
    __ADD_SUPP(YX);
    __ADD_SUPP(YY);
  }
  atomicAdd(&grid[grid_point_u][grid_point_v], sumXX, sumXY, sumYX, sumYY);
}

template <
    bool is_half_gcf
  >
__inline__ __device__
// psfi must be initialized to 0s.
void psfiKernel_scatter_kernel_small(
    int max_supp
    // For full-size GCF should be passed 0-centered,
    // i.e. with 0-index in the middle
  , const complexd * gcf[]
  , complexd _psfi[]
  , const Pregridded uvo_shared[]
  , const int2 off_shared[]
  , int myU
  , int myV
  , int timesteps_x_channels
  , int grid_size
  ) {
  __ACC(complexd, psfi, grid_size);
  complexd sum = {0, 0};
  int
      grid_point_u = 0
    , grid_point_v = 0;

  for (int i = 0; i < timesteps_x_channels; i++) {
    int myConvU, myConvV, myGridU, myGridV, u, v;
    u = uvo_shared[i].u;
    v = uvo_shared[i].v;
    myConvU = myU - off_shared[i].x;
    if (myConvU < 0) myConvU += max_supp;
    myConvV = myV - off_shared[i].y;
    if (myConvV < 0) myConvV += max_supp;
    // This translates points by max_supp/2
    // returning them back to normal (they were translates by -max_supp/2 before)
    myGridU = u + myConvU;
    myGridV = v + myConvV;

    int supp = uvo_shared[i].gcf_layer_supp;

    complexd supportPixel;
    #define __layeroff myConvU * supp + myConvV
    if (is_half_gcf) {
      int index = uvo_shared[i].gcf_layer_index;
      // Negative index indicates that original w was mirrored
      // and we shall negate the index to obtain correct
      // offset *and* conjugate the result.
      if (index < 0) {
        supportPixel = gcf[-index][__layeroff];
        supportPixel.y = - supportPixel.y;
      } else {
        supportPixel = gcf[index][__layeroff];
      }
    } else {
        supportPixel = gcf[uvo_shared[i].gcf_layer_index][__layeroff];
    }

    if (myGridU != grid_point_u || myGridV != grid_point_v) {
      atomicAdd(&psfi[grid_point_u][grid_point_v], sum);
      sum = make_hipDoubleComplex(0.0, 0.0);
      grid_point_u = myGridU;
      grid_point_v = myGridV;
    }
    sum = hipCadd(supportPixel, sum);
  }
  atomicAdd(&psfi[grid_point_u][grid_point_v], sum);
}

template <
    bool is_half_gcf
  >
__inline__ __device__
// grid must be initialized to 0s.
void gridKernel_scatter_kernel(
    int max_supp
  , const complexd * gcf[]
  , Double4c grid[]
  , const Pregridded uvo_shared[]
  , const Double4c vis_shared[]
  , const int2 off_shared[]
  , int timesteps_x_channels
  , int grid_size
  ) {
  for (int i = threadIdx.x; i < max_supp * max_supp; i += blockDim.x) {
    int
        myU = i % max_supp
      , myV = i / max_supp
      ;
    gridKernel_scatter_kernel_small<
      is_half_gcf
    > (max_supp, gcf, grid, uvo_shared, vis_shared, off_shared, myU, myV, timesteps_x_channels, grid_size);
  }
}

template <
    bool is_half_gcf
  >
__inline__ __device__
// grid must be initialized to 0s.
void psfiKernel_scatter_kernel(
    int max_supp
  , const complexd * gcf[]
  , complexd psfi[]
  , const Pregridded uvo_shared[]
  , const int2 off_shared[]
  , int timesteps_x_channels
  , int grid_size
  ) {
  for (int i = threadIdx.x; i < max_supp * max_supp; i += blockDim.x) {
    int
        myU = i % max_supp
      , myV = i / max_supp
      ;
    psfiKernel_scatter_kernel_small<
      is_half_gcf
    > (max_supp, gcf, psfi, uvo_shared, off_shared, myU, myV, timesteps_x_channels, grid_size);
  }
}

template <
    int over
  , bool is_half_gcf
  >
__device__ __inline__ void addBaselineToGrid(
    double scale
  , double wstep
  , int max_supp
  , Double4c grid[]
  , const complexd * gcf[]
  , const double3 uvw[]
  , const Double4c vis[]
  , int timesteps_x_channels
  , int grid_size
  ) {
  // NOTE: Don't forget to put timesteps_x_channels*80
  //   to kernel launch shared memory config.
  extern __shared__ Double4c vis_shared[];
  Pregridded * uvo_shared = reinterpret_cast<Pregridded *>(vis_shared + timesteps_x_channels);
  int2 * off_shared =  reinterpret_cast<int2 *>(uvo_shared + timesteps_x_channels);
  
  loadUVWIntoSharedMem<
      over
    , is_half_gcf
    >(scale
    , wstep
    , max_supp
    , uvw
    , uvo_shared
    , off_shared
    , timesteps_x_channels
    , grid_size
    );
  loadVisIntoSharedMem(
      vis
    , uvw
    , vis_shared
    , timesteps_x_channels
    );
  syncthreads();
  gridKernel_scatter_kernel<
      is_half_gcf
    >(max_supp
    , gcf
    , grid
    , uvo_shared
    , vis_shared
    , off_shared
    , timesteps_x_channels
    , grid_size
  );
}

template <
    int over
  , bool is_half_gcf
  >
__device__ __inline__ void addBaselineToPsfi(
    double scale
  , double wstep
  , int max_supp
  , complexd psfi[]
  , const complexd * gcf[]
  , const double3 uvw[]
  , int timesteps_x_channels
  , int grid_size
  ) {
  // NOTE: Don't forget to put timesteps_x_channels*16
  //   to kernel launch shared memory config.
  extern __shared__ Pregridded uvo_shared[];
  int2 * off_shared =  reinterpret_cast<int2 *>(uvo_shared + timesteps_x_channels);
  
  loadUVWIntoSharedMem<
      over
    , is_half_gcf
    >(scale
    , wstep
    , max_supp
    , uvw
    , uvo_shared
    , off_shared
    , timesteps_x_channels
    , grid_size
    );
  syncthreads();
  psfiKernel_scatter_kernel<
      is_half_gcf
    >(max_supp
    , gcf
    , psfi
    , uvo_shared
    , off_shared
    , timesteps_x_channels
    , grid_size
  );
}

template <
    int over
  , bool is_half_gcf
  , bool use_permutations
  >
__device__ __inline__ void addBaselinesToGrid(
    double scale
  , double wstep
  , const BlWMap permutations[/* baselines */]
  , Double4c grid[]
  , const complexd * gcf[]
  , const double3 uvw[]
  , const Double4c vis[]
  , int blOff
  , int timesteps_x_channels
  , int grid_size
  ) {
  int bl = blockIdx.x + blOff;
  if (use_permutations) bl = permutations[bl].bl;
  int max_supp = get_supp(permutations[bl].wp);

  addBaselineToGrid<
      over
    , is_half_gcf
    >(scale
    , wstep
    , max_supp
    , grid
    , gcf
    , uvw + bl * timesteps_x_channels
    , vis + bl * timesteps_x_channels
    , timesteps_x_channels
    , grid_size
    );
}

template <
    int over
  , bool is_half_gcf
  , bool use_permutations
  >
__device__ __inline__ void addBaselinesToPsfi(
    double scale
  , double wstep
  , const BlWMap permutations[/* baselines */]
  , complexd psfi[]
  , const complexd * gcf[]
  , const double3 uvw[]
  , int blOff
  , int timesteps_x_channels
  , int grid_size
  ) {
  int bl = blockIdx.x + blOff;
  if (use_permutations) bl = permutations[bl].bl;
  int max_supp = get_supp(permutations[bl].wp);

  addBaselineToPsfi<
      over
    , is_half_gcf
    >(scale
    , wstep
    , max_supp
    , psfi
    , gcf
    , uvw + bl * timesteps_x_channels
    , timesteps_x_channels
    , grid_size
    );
}

#define addBaselineToGrid(suff, ishalf)  \
extern "C"                               \
__global__ void addBaselineToGrid##suff( \
    double scale                         \
  , double wstep                         \
  , int max_supp                         \
  , Double4c grid[]                      \
  , const complexd * gcf[]               \
  , const double3 uvw[]                  \
  , const Double4c vis[]                 \
  , int timesteps_x_channels             \
  , int grid_size                        \
  ) {                                    \
  addBaselineToGrid<OVER, ishalf>        \
    ( scale                              \
    , wstep                              \
    , max_supp                           \
    , grid                               \
    , gcf                                \
    , uvw                                \
    , vis                                \
    , timesteps_x_channels               \
    , grid_size                          \
    );                                   \
}
addBaselineToGrid(HalfGCF, true)
addBaselineToGrid(FullGCF, false)

#define addBaselinesToGridSkaMid(suff, ishalf)  \
extern "C"                                      \
__global__ void addBaselinesToGridSkaMid##suff( \
    double scale                                \
  , double wstep                                \
  , const BlWMap permutations[/* baselines */]  \
  , Double4c grid[]                             \
  , const complexd * gcf[]                      \
  , const double3 uvw[]                         \
  , const Double4c vis[]                        \
  , int blOff                                   \
  , int timesteps_x_channels                    \
  , int grid_size                               \
  ) {                                           \
  addBaselinesToGrid<OVER, ishalf, false>       \
    ( scale                                     \
    , wstep                                     \
    , permutations                              \
    , grid                                      \
    , gcf                                       \
    , uvw                                       \
    , vis                                       \
    , blOff                                     \
    , timesteps_x_channels                      \
    , grid_size                                 \
    );                                          \
}
addBaselinesToGridSkaMid(HalfGCF, true)
addBaselinesToGridSkaMid(FullGCF, false)

#define addBaselinesToGridSkaMidUsingPermutations(suff, ishalf)  \
extern "C"                                                       \
__global__ void addBaselinesToGridSkaMidUsingPermutations##suff( \
    double scale                                                 \
  , double wstep                                                 \
  , const BlWMap permutations[/* baselines */]                   \
  , Double4c grid[]                                              \
  , const complexd * gcf[]                                       \
  , const double3 uvw[]                                          \
  , const Double4c vis[]                                         \
  , int blOff                                                    \
  , int timesteps_x_channels                                     \
  , int grid_size                                                \
  ) {                                                            \
  addBaselinesToGrid<OVER, ishalf, true>                         \
    ( scale                                                      \
    , wstep                                                      \
    , permutations                                               \
    , grid                                                       \
    , gcf                                                        \
    , uvw                                                        \
    , vis                                                        \
    , blOff                                                      \
    , timesteps_x_channels                                       \
    , grid_size                                                  \
    );                                                           \
}
addBaselinesToGridSkaMidUsingPermutations(HalfGCF, true)
addBaselinesToGridSkaMidUsingPermutations(FullGCF, false)

#define addBaselinesToPsfiSkaMid(suff, ishalf)       \
extern "C"                                           \
__global__ void addBaselinesToPsfiSkaMid##suff(      \
    double scale                                     \
  , double wstep                                     \
  , const BlWMap permutations[/* baselines */]       \
  , complexd psfi[]                                  \
  , const complexd * gcf[]                           \
  , const double3 uvw[]                              \
  , int blOff                                        \
  , int timesteps_x_channels                         \
  , int grid_size                                    \
  ) {                                                \
  addBaselinesToPsfi<OVER, ishalf, false>            \
    ( scale                                          \
    , wstep                                          \
    , permutations                                   \
    , psfi                                           \
    , gcf                                            \
    , uvw                                            \
    , blOff                                          \
    , timesteps_x_channels                           \
    , grid_size                                      \
    );                                               \
}
addBaselinesToPsfiSkaMid(HalfGCF, true)
addBaselinesToPsfiSkaMid(FullGCF, false)

#define addBaselinesToPsfiSkaMidUsingPermutations(suff, ishalf)  \
extern "C"                                                       \
__global__ void addBaselinesToPsfiSkaMidUsingPermutations##suff( \
    double scale                                                 \
  , double wstep                                                 \
  , const BlWMap permutations[/* baselines */]                   \
  , complexd psfi[]                                              \
  , const complexd * gcf[]                                       \
  , const double3 uvw[]                                          \
  , int blOff                                                    \
  , int timesteps_x_channels                                     \
  , int grid_size                                                \
  ) {                                                            \
  addBaselinesToPsfi<OVER, ishalf, true>                         \
    ( scale                                                      \
    , wstep                                                      \
    , permutations                                               \
    , psfi                                                       \
    , gcf                                                        \
    , uvw                                                        \
    , blOff                                                      \
    , timesteps_x_channels                                       \
    , grid_size                                                  \
    );                                                           \
}
addBaselinesToPsfiSkaMidUsingPermutations(HalfGCF, true)
addBaselinesToPsfiSkaMidUsingPermutations(FullGCF, false)

#include "../GCF_new/scale_complex_by_dbl.cuh"

typedef complexd poltyp[4];

extern "C"
__global__ void  normalizeAndExtractPolarization_dyn(
    complexd _dst_grid[]
  , const poltyp _src_grid[]
  , int pol
  , int grid_size
  , double inv_grid_size_2 // 1/(grid_size^2)
  )
{
  __ACC(complexd, dst_grid, grid_size);
  __ACC(poltyp, src_grid, grid_size);
  const int
      x = blockIdx.x * blockDim.x + threadIdx.x
    , y = blockIdx.y * blockDim.y + threadIdx.y;

  dst_grid[x][y] = cuMulComplexByDouble(src_grid[x][y][pol], inv_grid_size_2);
}
