#include "hip/hip_runtime.h"
#include "common.h"
#include "metrix.h"
#include "atomic_add.h"
#include "OskarBinReader.h"

#include <math_functions.h>

template <
    int over
  , int w_planes
  , int grid_size

  , int timesteps
  , int channels
  , bool do_mirror
  >
__inline__ __device__ void loadIntoSharedMem (
    double scale
  , double wstep
  , int max_supp // needed for u v translation
  , const double3 uvw[timesteps * channels]
  , const Double4c vis[timesteps * channels]
  , Pregridded uvo_shared[timesteps * channels]
  , Double4c vis_shared[timesteps * channels]
  ) {
  for (int i = threadIdx.x; i < timesteps * channels; i += blockDim.x) {
    // uvo_shared[i] is passed by reference and updated!
    pregridPoint<grid_size, over, w_planes, do_mirror>(scale, wstep, uvw[i], uvo_shared[i]);
    vis_shared[i] = vis[i];
  }
}

template <
    int grid_size
  , int w_planes

  , int timesteps
  , int channels
  , bool is_half_gcf
  >
__inline__ __device__
// grid must be initialized to 0s.
void gridKernel_scatter_kernel_small(
    int max_supp
    // For full-size GCF should be passed 0-centered,
    // i.e. with 0-index in the middle
  , const complexd * gcf[]
  , Double4c grid[grid_size][grid_size]
  , const Pregridded uvo_shared[timesteps * channels]
  , const Double4c vis[timesteps * channels]
  , int myU
  , int myV
  ) {
  complexd
      sumXX = {0, 0}
    , sumXY = {0, 0}
    , sumYX = {0, 0}
    , sumYY = {0, 0}
    ;
  int
      grid_point_u = 0
    , grid_point_v = 0;

  for (int i = 0; i < timesteps * channels; i++) {
    int myConvU, myConvV, myGridU, myGridV, u, v;
    u = uvo_shared[i].u;
    v = uvo_shared[i].v;
    myConvU = myU - u % max_supp;
    if (myConvU < 0) myConvU += max_supp;
    myConvV = myV - v % max_supp;
    if (myConvV < 0) myConvV += max_supp;
    // This translates points by max_supp/2
    // returning them back to normal (they were translates by -max_supp/2 before)
    myGridU = u + myConvU;
    myGridV = v + myConvV;

    int supp = uvo_shared[i].gcf_layer_supp;

    complexd supportPixel;
    #define __layeroff myConvU * supp + myConvV
    if (is_half_gcf) {
      int index = uvo_shared[i].gcf_layer_index;
      // Negative index indicates that original w was mirrored
      // and we shall negate the index to obtain correct
      // offset *and* conjugate the result.
      if (index < 0) {
        supportPixel = gcf[-index][__layeroff];
        supportPixel.y = - supportPixel.y;
      } else {
        supportPixel = gcf[index][__layeroff];
      }
    } else {
        supportPixel = gcf[uvo_shared[i].gcf_layer_index][__layeroff];
    }

    if (myGridU != grid_point_u || myGridV != grid_point_v) {
      atomicAdd(&grid[grid_point_u][grid_point_v], sumXX, sumXY, sumYX, sumYY);
        sumXX
      = sumXY
      = sumYX
      = sumYY
      = make_hipDoubleComplex(0.0, 0.0);
      grid_point_u = myGridU;
      grid_point_v = myGridV;
    }
    #define __ADD_SUPP(pol) sum##pol = hipCfma(supportPixel, vis[i].pol, sum##pol)
    __ADD_SUPP(XX);
    __ADD_SUPP(XY);
    __ADD_SUPP(YX);
    __ADD_SUPP(YY);
  }
  atomicAdd(&grid[grid_point_u][grid_point_v], sumXX, sumXY, sumYX, sumYY);
}

template <
    int grid_size
  , int w_planes

  , int timesteps
  , int channels
  , bool is_half_gcf
  >
__inline__ __device__
// grid must be initialized to 0s.
void gridKernel_scatter_kernel(
    int max_supp
  , const complexd * gcf[]
  , Double4c grid[grid_size][grid_size]
  , const Pregridded uvo_shared[timesteps * channels]
  , const Double4c vis[timesteps * channels]
  ) {
  for (int i = threadIdx.x; i < max_supp * max_supp; i += blockDim.x) {
    int
        myU = i % max_supp
      , myV = i / max_supp
      ;
    gridKernel_scatter_kernel_small<
      grid_size
    , w_planes

    , timesteps
    , channels
    , is_half_gcf
    > (max_supp, gcf, grid, uvo_shared, vis, myU, myV);
  }
}

template <
    int over
  , int w_planes
  , int grid_size

  , int timesteps
  , int channels
  , bool is_half_gcf
  >
__device__ __inline__ void addBaselineToGrid(
    double scale
  , double wstep
  , int max_supp
  , Double4c grid[grid_size][grid_size]
  , const complexd * gcf[]
  , const double3 uvw[timesteps * channels]
  , const Double4c vis[timesteps * channels]
  ) {
  __shared__ Pregridded uvo_shared[timesteps * channels];
  __shared__ Double4c vis_shared[timesteps * channels];
  
  loadIntoSharedMem<
      over
    , w_planes
    , grid_size
    , timesteps
    , channels
    , is_half_gcf
    >(scale
    , wstep
    , max_supp
    , uvw
    , vis
    , uvo_shared
    , vis_shared
    );
  syncthreads();
  gridKernel_scatter_kernel<
      grid_size
    , w_planes
    , timesteps
    , channels
    , is_half_gcf
    >(max_supp
    , gcf
    , grid
    , uvo_shared
    , vis_shared
  );
}

template <
    int over
  , int w_planes
  , int grid_size

  , int baselines
  , int timesteps
  , int channels
  , bool is_half_gcf
  , bool use_permutations
  >
__device__ __inline__ void addBaselinesToGrid(
    double scale
  , double wstep
  , const BlWMap permutations[baselines]
  , Double4c grid[grid_size][grid_size]
  , const complexd * gcf[]
  , const double3 uvw[baselines][timesteps * channels]
  , const Double4c vis[baselines][timesteps * channels]
  ) {
  int bl = blockIdx.x;
  if (use_permutations) bl = permutations[blockIdx.x].bl;
  int max_supp = get_supp(permutations[bl].wp);

  addBaselineToGrid<
      over
    , w_planes
    , grid_size
    , timesteps
    , channels
    , is_half_gcf
    >(scale
    , wstep
    , max_supp
    , grid
    , gcf
    , uvw[bl]
    , vis[bl]
    );
}

#define addBaselineToGrid(suff, ishalf)                                    \
extern "C"                                                                 \
__global__ void addBaselineToGrid##suff(                                   \
    double scale                                                           \
  , double wstep                                                           \
  , int max_supp                                                           \
  , Double4c grid[GRID_SIZE][GRID_SIZE]                                    \
  , const complexd * gcf[]                                                 \
  , const double3 uvw[TIMESTEPS*CHANNELS]                                  \
  , const Double4c vis[TIMESTEPS*CHANNELS]                                 \
  ) {                                                                      \
  addBaselineToGrid<OVER, WPLANES, GRID_SIZE, TIMESTEPS, CHANNELS, ishalf> \
    ( scale                                                                \
    , wstep                                                                \
    , max_supp                                                             \
    , grid                                                                 \
    , gcf                                                                  \
    , uvw                                                                  \
    , vis                                                                  \
    );                                                                     \
}
addBaselineToGrid(HalfGCF, true)
addBaselineToGrid(FullGCF, false)

#define addBaselinesToGridSkaMid(suff, ishalf)                                                \
extern "C"                                                                                    \
__global__ void addBaselinesToGridSkaMid##suff(                                               \
    double scale                                                                              \
  , double wstep                                                                              \
  , const BlWMap permutations[BASELINES]                                                      \
  , Double4c grid[GRID_SIZE][GRID_SIZE]                                                       \
  , const complexd * gcf[]                                                                    \
  , const double3 uvw[BASELINES][TIMESTEPS*CHANNELS]                                          \
  , const Double4c vis[BASELINES][TIMESTEPS*CHANNELS]                                         \
  ) {                                                                                         \
  addBaselinesToGrid<OVER, WPLANES, GRID_SIZE, BASELINES, TIMESTEPS, CHANNELS, ishalf, false> \
    ( scale                                                                                   \
    , wstep                                                                                   \
    , permutations                                                                            \
    , grid                                                                                    \
    , gcf                                                                                     \
    , uvw                                                                                     \
    , vis                                                                                     \
    );                                                                                        \
}
addBaselinesToGridSkaMid(HalfGCF, true)
addBaselinesToGridSkaMid(FullGCF, false)

#define addBaselinesToGridSkaMidUsingPermutations(suff, ishalf)                              \
extern "C"                                                                                   \
__global__ void addBaselinesToGridSkaMidUsingPermutations##suff(                             \
    double scale                                                                             \
  , double wstep                                                                             \
  , const BlWMap permutations[BASELINES]                                                     \
  , Double4c grid[GRID_SIZE][GRID_SIZE]                                                      \
  , const complexd * gcf[]                                                                   \
  , const double3 uvw[BASELINES][TIMESTEPS*CHANNELS]                                         \
  , const Double4c vis[BASELINES][TIMESTEPS*CHANNELS]                                        \
  ) {                                                                                        \
  addBaselinesToGrid<OVER, WPLANES, GRID_SIZE, BASELINES, TIMESTEPS, CHANNELS, ishalf, true> \
    ( scale                                                                                  \
    , wstep                                                                                  \
    , permutations                                                                           \
    , grid                                                                                   \
    , gcf                                                                                    \
    , uvw                                                                                    \
    , vis                                                                                    \
    );                                                                                       \
}
addBaselinesToGridSkaMidUsingPermutations(HalfGCF, true)
addBaselinesToGridSkaMidUsingPermutations(FullGCF, false)

#include "../GCF_new/scale_complex_by_dbl.cuh"

extern "C"
__global__ void  normalizeAndExtractPolarization(
    int pol
  , complexd dst_grid[GRID_SIZE][GRID_SIZE]
  , const complexd src_grid[GRID_SIZE][GRID_SIZE][4]
  )
{
  const int
      x = blockIdx.x * blockDim.x + threadIdx.x
    , y = blockIdx.y * blockDim.y + threadIdx.y;

  dst_grid[x][y] = cuMulComplexByDouble(src_grid[x][y][pol], 1.0/(GRID_SIZE*GRID_SIZE));
}
