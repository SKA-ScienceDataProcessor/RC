#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

// Launch configuration should be as follows:
//  1. blocks of dim3(threads_per_dim, threads_per_dim, 1) size
//       where threads_per_dim = min(N, 16)
//  2. grid of dim3((N+threads_per_dim-1)/threads_per_dim, (N-1)/(threads_per_dim * 2)+1, 1) blocks

template <class T>
static __device__ __inline__
void fftshift_kernel_common(T* data, int shift, int N, int pitch){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x > N-1 || y > (N+1)/2-1) return;

  int x1 = x + shift;
  if (x1 > N-1) x1 -= N;

  int y1 = y + shift;
  if (y1 > N-1) y1 -= N;

  int
      i = x + y * pitch
    , i1 = x1 + y1 * pitch;

  T tmp;
  tmp = data[i];
  data[i] = data[i1];
  data[i1] = tmp;
}

extern "C" {
__global__
void fft_make_hermitian_kernel(hipDoubleComplex* data, int N, int pitch) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= N/2 || y >= N) return;
  // Add conjugated value from other side
  int i = x+y*pitch,
      i1 = (N-1)*(pitch+1)-i; // (N-x-1)+(N-y-1)*pitch;
  data[i].x += data[i1].x;
  data[i].y -= data[i1].y;
  // cuFFT wants floor(N/2)+1 valid lines. So for the last line we
  // actually need to update the other side of the grid, too.
  if (x+1 >= N/2) {
    data[i1].x = data[i].x;
    data[i1].y = -data[i].y;
  }
}

__global__
void fftshift_kernel_cx(hipDoubleComplex* data, int N, int pitch) {
  fftshift_kernel_common<hipDoubleComplex>(data, N/2, N, pitch);
}

__global__
void ifftshift_kernel_cx(hipDoubleComplex* data, int N, int pitch) {
  fftshift_kernel_common<hipDoubleComplex>(data, N/2 + N%2, N, pitch);
}

__global__
void fftshift_kernel_r(double* data, int N, int pitch) {
  fftshift_kernel_common<double>(data, N/2, N, pitch);
}

__global__
void ifftshift_kernel_r(double* data, int N, int pitch) {
  fftshift_kernel_common<double>(data, N/2 + N%2, N, pitch);
}
}
