#include "hip/hip_runtime.h"

#include "threadFenceReduction_kernel.cuh"

struct doubleSum {
  TASKCFG double init() {return 0.0;}
  TASKCFG double reduce(double x, double acc){return x + acc;}
  TASKCFG double f(unsigned int, hipDoubleComplex c){return c.x;}
};

template <unsigned int blockSize, bool nIsPow2>
TASKCFG
void reduceSinglePass_dev(const hipDoubleComplex *g_idata, double *g_odata, unsigned int n){
  reduceSinglePass_devGen<blockSize, nIsPow2, double, hipDoubleComplex, doubleSum>(g_idata, g_odata, n);
}

extern "C" __host__
void reduce_init() {
    resetRetirementCount();
}

extern "C" __global__ void reduce_512_e2(const hipDoubleComplex *g_idata, double *g_odata, unsigned int n) {
  reduceSinglePass_dev<512, true>(g_idata, g_odata, n);
}

__device__ static __inline__ hipDoubleComplex cuMulComplexByDouble(hipDoubleComplex v,
                                                             double y){
  return make_hipDoubleComplex ( v.x * y
                              , v.y * y
                              );
}

__device__ __inline__
void scale_complex_by_dbl(
    double norm
  , hipDoubleComplex * v
  , int len
  ) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < len) v[x] = cuMulComplexByDouble(v[x], norm);
}

extern "C" __global__ void normalize(
    double * normp
  , hipDoubleComplex * v
  , int len
  ){
  scale_complex_by_dbl(1.0 / (*normp), v, len);
}

// Helper for calling hipConfigureCall from Haskell-land
extern "C" hipError_t __cudaConfigureCall
(
    int gridX,  int gridY, int gridZ,
    int blockX, int blockY, int blockZ,
    size_t sharedMem,
    hipStream_t stream
)
{
    dim3 gridDim(gridX, gridY, gridZ);
    dim3 blockDim(blockX,blockY,blockZ);

    return hipConfigureCall(gridDim, blockDim, sharedMem, stream);
}
